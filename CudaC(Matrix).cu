#include "hip/hip_runtime.h"
%%writefile cuda.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void initialise_Matrix(int *a, int n, int m){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++){
            a[i*m+j] = rand() % 100;
        }
    }
}

void print(int* a, int num, int col){
    for(int i = 0; i < num; i++){
        for(int j = 0; j < col; j++){
            printf("%d ", a[i*col + j]);
        }
        printf("\n");
    }
}

__global__ void matrixMultiplicationCuda (int *a, int *b, int *c, int n, int m, int p){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    int temp = 0;

    if((row < n) && (column < p)){
        for(int i = 0; i < m; i++){
            temp += a[row*m+i] * b[i*p+column];
        }
        c[row * p + column] = temp;
    }
}

int main(){
    int num = 1000;
    int col = 2000;
    int p = 1000;

    size_t bytes = num*col*sizeof(int);

    int *host_A, *host_B, *host_C;

    host_A = (int*) malloc(bytes);
    host_B = (int*) malloc(bytes);
    host_C = (int*) malloc(bytes);

    int *Device_A, *Device_B, *Device_C;

    hipMalloc(&Device_A, bytes);
    hipMalloc(&Device_B, bytes);
    hipMalloc(&Device_C, bytes);

    initialise_Matrix(host_A, num, col);
    initialise_Matrix(host_B, col, p);

    hipMemcpy(Device_A, host_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(Device_B, host_B, bytes, hipMemcpyHostToDevice);

    int BLOCK_SIZE = 16;
    int GRID_SIZE = (int)ceil(num/ BLOCK_SIZE);

    dim3 grid(GRID_SIZE,GRID_SIZE);
    dim3 threads(BLOCK_SIZE,BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, NULL);

    matrixMultiplicationCuda <<<grid, threads>>> (Device_A, Device_B, Device_C, num, col, p);

    hipMemcpy(host_C, Device_C, bytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The elapsed time in gpu was %.2f ms\n", milliseconds);
    printf("The number of threads: %d\n", GRID_SIZE * GRID_SIZE * BLOCK_SIZE * BLOCK_SIZE);

    print(host_C, num, p);

    hipFree(Device_A);
    hipFree(Device_B);
    hipFree(Device_C);
    free(host_A);
    free(host_B);
    free(host_C);

    return 0;
}
